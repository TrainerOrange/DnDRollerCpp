
#include <hip/hip_runtime.h>
#include <cstdio>
#include <ctime>

/* we need these includes for CUDA's random number stuff */
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <random>
#include <chrono>
#include "hip/hip_runtime_api.h"

#define N 8
#define MAX 20
#define PERCENTAGEINTERVAL 5

/* this GPU kernel function is used to initialize the random states */
__global__ void init(unsigned int seed, hiprandState_t* states) {

    /* we have to initialize the state */
    hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
                blockIdx.x, /* the sequence number should be different for each core (unless you want all
                             cores to get the same sequence of numbers for some reason - use thread id! */
                blockIdx.y, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
                &states[blockIdx.x]);
}

/* this GPU kernel takes an array of states, and an array of ints, and puts a random int into each */
__global__ void randoms(hiprandState_t* states, uint8_t* numbers) {
    /* curand works like rand - except that it takes a state as a parameter */
    uint32_t randInt = hiprand(&states[blockIdx.x]);
    numbers[4 * blockIdx.x]     = ((randInt & 0xFF000000UL) >> 24)  % MAX + 1;
    numbers[4 * blockIdx.x + 1] = ((randInt & 0x00FF0000UL) >> 16)  % MAX + 1;
    numbers[4 * blockIdx.x + 2] = ((randInt & 0x0000FF00UL) >> 8 )  % MAX + 1;
    numbers[4 * blockIdx.x + 3] = ((randInt & 0x000000FFUL)      )  % MAX + 1;
}

/* this GPU kernel takes an array of ints and adds 1 to the passcounter if they're greater than or equal to the given int */
__global__ void passcheck(unsigned long long int* passcounter, int8_t* numberstopass, const uint8_t* numbers) {
    if (numbers[blockIdx.x * blockDim.x + threadIdx.x] >= numberstopass[0]) { atomicAdd(passcounter, 1); }
}

void printLoadingBar(long long int rolled, long long int counterStop, double start_time ) {
    printf("Rolled: %lld%% ", (long long int)ceil(rolled * 100/counterStop));
    auto end = std::chrono::system_clock::now().time_since_epoch().count();
    double diff = end - start_time;
    printf(": %lld rolls per second \n", (long long int)ceil(((double)rolled * 10000000) / diff));
}

// you must first call the cudaGetDeviceProperties() function, then pass
// the devProp structure returned to this function:
int getSPcores(hipDeviceProp_t devProp)
{
    int cores = 0;
    int mp = devProp.multiProcessorCount;
    switch (devProp.major){
        case 2: // Fermi
            if (devProp.minor == 1) cores = mp * 48;
            else cores = mp * 32;
            break;
        case 3: // Kepler
            cores = mp * 192;
            break;
        case 5: // Maxwell
            cores = mp * 128;
            break;
        case 6: // Pascal
            if ((devProp.minor == 1) || (devProp.minor == 2)) cores = mp * 128;
            else if (devProp.minor == 0) cores = mp * 64;
            else printf("Unknown device type\n");
            break;
        case 7: // Volta and Turing
            if ((devProp.minor == 0) || (devProp.minor == 5)) cores = mp * 64;
            else printf("Unknown device type\n");
            break;
        case 8: // Ampere
            if (devProp.minor == 0) cores = mp * 64;
            else if (devProp.minor == 6) cores = mp * 128;
            else printf("Unknown device type\n");
            break;
        default:
            printf("Unknown device type\n");
            break;
    }
    return cores;
}

int main() {
    hipDeviceProp_t hipDeviceProp_t;
    hipGetDeviceProperties(&hipDeviceProp_t, 0);
    int nrCores = getSPcores(hipDeviceProp_t);
    std::cout << "ShaderCores: " << nrCores << "\n";

    // After how many rolls should you stop
    long long int counter = 0;
    long long int counterstop = (long long int)(INT32_MAX) * 2;

    // Cuda performance metrics
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

//    /* allocate an array of int8_t on the CPU and GPU */
//    uint8_t cpu_nums[ 4];
    uint8_t* gpu_nums;
    hipMalloc((void **) &gpu_nums, nrCores * N * 4 * sizeof(uint8_t));

    /* allocate an array of int8_t on the CPU and GPU */
    unsigned long long int cpu_pass_counter[1];
    cpu_pass_counter[0] = 0;
    unsigned long long int* gpu_pass_counter;
    hipMalloc((void **) &gpu_pass_counter, 1 * sizeof(unsigned long long int));
    hipMemcpy(gpu_pass_counter, cpu_pass_counter, 1 * sizeof(unsigned long long int), hipMemcpyHostToDevice);

    /* allocate an array of int8_t on the CPU and GPU of numbers that should be checked against */
    int8_t cpu_num_to_pass[1];
    cpu_num_to_pass[0] = 11;
    int8_t* gpu_num_to_roll;
    hipMalloc((void **) &gpu_num_to_roll, 1 * sizeof(int8_t));
    hipMemcpy(gpu_num_to_roll, cpu_num_to_pass, 1 * sizeof(int8_t), hipMemcpyHostToDevice);

    hipDeviceSynchronize();

    /* allocate space on the GPU for the random states */
    hiprandState_t* states;
    hipMalloc((void **) &states, nrCores * N * sizeof(hiprandState_t));

    /* invoke the GPU to initialize all of the random states */
    init<<<nrCores * N, 1>>>(time(nullptr), states);

    auto start_timer = std::chrono::system_clock::now();
    printLoadingBar(counter, counterstop, start_timer.time_since_epoch().count());
    hipEventRecord(start);
    int loopcounter = 0;
    while (counter < counterstop) {
        /* invoke the kernel to get some random numbers */
        randoms<<<nrCores * N, 1>>>(states, gpu_nums);
        passcheck<<<nrCores * N, 4>>>(gpu_pass_counter, gpu_num_to_roll, gpu_nums);

        /* copy the random numbers back */
//        cudaMemcpy(cpu_nums, gpu_nums, nrCores * N * 4 * sizeof(int8_t), cudaMemcpyDeviceToHost);
//        cudaMemcpy(cpu_pass_counter, gpu_pass_counter, 1 * sizeof(int64_t), cudaMemcpyDeviceToHost);

        counter += nrCores * N * 4;
        if ((loopcounter % ((counterstop/(N * nrCores * 4)) / (int)( 1 / ((float)(PERCENTAGEINTERVAL) / 100 ) ))) == 0) {
            printLoadingBar(counter, counterstop, start_timer.time_since_epoch().count());
        }
        loopcounter++;
    }
    hipEventRecord(stop);
    printLoadingBar(counter, counterstop, start_timer.time_since_epoch().count());
    hipMemcpy(cpu_pass_counter, gpu_pass_counter, 1 * sizeof(unsigned long long int), hipMemcpyDeviceToHost);

    /* free memory from GPU */
    hipFree(states);
    hipFree(gpu_nums);
    hipFree(gpu_pass_counter);
    hipFree(gpu_num_to_roll);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Ran %lld simulations resulting in %lld d%i rolls above %i taking %fs \n", counter, cpu_pass_counter[0], MAX, cpu_num_to_pass[0], milliseconds/1000);
    printf("Averaged: %lld rolls per second", (long long int)(counter / (milliseconds/1000)));

    do
    {
        std::cout << '\n' << "Enter any key to continue...";
    } while (std::cin.get() != '\n');

    return 0;
}


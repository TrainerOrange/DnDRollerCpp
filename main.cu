
#include <hip/hip_runtime.h>
#include <cstdio>
#include <ctime>

/* we need these includes for CUDA's random number stuff */
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <random>
#include <chrono>
#include <ctime>

#define N 8704
#define R 1
#define MAX 20

/* this GPU kernel function is used to initialize the random states */
__global__ void init(unsigned int seed, hiprandState_t* states) {

    /* we have to initialize the state */
    hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
                blockIdx.x, /* the sequence number should be different for each core (unless you want all
                             cores to get the same sequence of numbers for some reason - use thread id! */
                blockIdx.y, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
                &states[blockIdx.x]);
}

/* this GPU kernel takes an array of states, and an array of ints, and puts a random int into each */
__global__ void randoms(hiprandState_t* states, uint8_t* numbers) {
    /* curand works like rand - except that it takes a state as a parameter */
    uint32_t randInt = hiprand(&states[blockIdx.x]);
    numbers[4 * blockIdx.x]     = ((randInt & 0xFF000000UL) >> 24)  % MAX + 1;
    numbers[4 * blockIdx.x + 1] = ((randInt & 0x00FF0000UL) >> 16)  % MAX + 1;
    numbers[4 * blockIdx.x + 2] = ((randInt & 0x0000FF00UL) >> 8 )  % MAX + 1;
    numbers[4 * blockIdx.x + 3] = ((randInt & 0x000000FFUL)      )  % MAX + 1;
}

/* this GPU kernel takes an array of ints and adds 1 to the passcounter if they're greater than or equal to the given int */
__global__ void passcheck(unsigned long long int* passcounter, int8_t* numberstopass, const uint8_t* numbers) {
    if (numbers[blockIdx.x * blockDim.x + threadIdx.x] >= numberstopass[0]) { atomicAdd(passcounter, 1); }
}

void printLoadingBar(unsigned long long int rolled, unsigned long long int counterStop, double start_time ) {
    printf("Rolled: %lld%% ", (rolled * 100)/counterStop);
    auto end = std::chrono::system_clock::now().time_since_epoch().count();
    double diff = end - start_time;
    printf(": %i rolls per second \n", (int)(((double)rolled * 10000000) / diff));
}

int main() {
    // After how many rolls should you stop
    unsigned long long int counter = 0;
    unsigned long long int counterstop = (unsigned long long int)(INT32_MAX/512) * N;

    // Cuda performance metrics
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

//    /* allocate an array of int8_t on the CPU and GPU */
//    uint8_t cpu_nums[N * 4];
    uint8_t* gpu_nums;
    hipMalloc((void **) &gpu_nums, N * 4 * sizeof(uint8_t));

    /* allocate an array of int8_t on the CPU and GPU */
    unsigned long long int cpu_pass_counter[1];
    cpu_pass_counter[0] = 0;
    unsigned long long int* gpu_pass_counter;
    hipMalloc((void **) &gpu_pass_counter, 1 * sizeof(unsigned long long int));
    hipMemcpy(gpu_pass_counter, cpu_pass_counter, 1 * sizeof(unsigned long long int), hipMemcpyHostToDevice);

    /* allocate an array of int8_t on the CPU and GPU of numbers that should be checked against */
    int8_t cpu_num_to_pass[R];
    cpu_num_to_pass[0] = 11;
    int8_t* gpu_num_to_roll;
    hipMalloc((void **) &gpu_num_to_roll, R * sizeof(int8_t));
    hipMemcpy(gpu_num_to_roll, cpu_num_to_pass, R * sizeof(int8_t), hipMemcpyHostToDevice);

    hipDeviceSynchronize();

    /* allocate space on the GPU for the random states */
    hiprandState_t* states;
    hipMalloc((void **) &states, N * sizeof(hiprandState_t));

    /* invoke the GPU to initialize all of the random states */
    init<<<N, 1>>>(time(nullptr), states);

    auto start_timer = std::chrono::system_clock::now();
    printLoadingBar(counter, counterstop, start_timer.time_since_epoch().count());
    hipEventRecord(start);
    while (counter < counterstop) {
        /* invoke the kernel to get some random numbers */
        randoms<<<N, 1>>>(states, gpu_nums);
        passcheck<<<N, 4>>>(gpu_pass_counter, gpu_num_to_roll, gpu_nums);

        /* copy the random numbers back */
//        cudaMemcpy(cpu_nums, gpu_nums, N * 4 * sizeof(int8_t), cudaMemcpyDeviceToHost);
//        cudaMemcpy(cpu_pass_counter, gpu_pass_counter, 1 * sizeof(int64_t), cudaMemcpyDeviceToHost);

        counter += N*4;
        if ((counter % ((N * 4) * 10000)) == 0) {
            printLoadingBar(counter, counterstop, start_timer.time_since_epoch().count());
        }
    }
    hipEventRecord(stop);
    printLoadingBar(counter, counterstop, start_timer.time_since_epoch().count());
    hipMemcpy(cpu_pass_counter, gpu_pass_counter, 1 * sizeof(unsigned long long int), hipMemcpyDeviceToHost);

    /* free memory from GPU */
    hipFree(states);
    hipFree(gpu_nums);
    hipFree(gpu_pass_counter);
    hipFree(gpu_num_to_roll);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Ran %lld simulations resulting in %lld passes taking %fs \n", counter, cpu_pass_counter[0], milliseconds/1000);
    printf("Averaged: %i rolls per second", (int)(counter/(milliseconds/1000)));

    return 0;
}

